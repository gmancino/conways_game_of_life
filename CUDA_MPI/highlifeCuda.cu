// Include packages and also CUDA packages
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>


// Result from last compute of world.
extern unsigned char *g_resultData;

// Current state of world. 
extern unsigned char *g_data;

// ----- SAVE RECEIVING ROWS FROM OTHER GPUS ----- //
// "Above" row
extern unsigned char *g_aboveRow;

// "Below" row 
extern unsigned char *g_belowRow;

// "Above" row
extern unsigned char *g_resultAboveRow;

// "Below" row 
extern unsigned char *g_resultBelowRow;

// ----- DECLARE KERNEL ----- //
__global__ void HL_kernel(unsigned int worldWidth, unsigned int worldHeight);


// Define number of Processors
int cudaDeviceCount;
hipError_t cE; 

static inline void HL_initAllZeros(size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    size_t total_world_size = worldWidth * worldHeight;

    // Initialize the data
    hipMallocManaged(&g_data, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_data, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the resulting data
    hipMallocManaged(&g_resultData, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the above row
    hipMallocManaged(&g_aboveRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_aboveRow, 0, (worldWidth * sizeof(unsigned char)));

    // Initialize the below row
    hipMallocManaged(&g_belowRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_belowRow, 0, (worldWidth * sizeof(unsigned char)));
}

static inline void HL_initAllOnes(size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    size_t total_world_size = worldWidth * worldHeight;

    // Initialize the data
    hipMallocManaged(&g_data, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_data, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the resulting data
    hipMallocManaged(&g_resultData, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the above row
    hipMallocManaged(&g_aboveRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_aboveRow, 0, (worldWidth * sizeof(unsigned char)));

    // Initialize the below row
    hipMallocManaged(&g_belowRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_belowRow, 0, (worldWidth * sizeof(unsigned char)));

    int i;

    // set all rows of world to true
    for( i = 0; i < total_world_size; i++)
    {
	    g_data[i] = 1;

        // Set above and below rows
        if (i < worldWidth){
            g_aboveRow[i] = 1;
            g_belowRow[i] = 1;
        }
    }
}

static inline void HL_initOnesInMiddle(size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    size_t total_world_size = worldWidth * worldHeight;

    // Initialize the data
    hipMallocManaged(&g_data, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_data, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the resulting data
    hipMallocManaged(&g_resultData, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the above row
    hipMallocManaged(&g_aboveRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_aboveRow, 0, (worldWidth * sizeof(unsigned char)));

    // Initialize the below row
    hipMallocManaged(&g_belowRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_belowRow, 0, (worldWidth * sizeof(unsigned char)));

    int i;
    for(i = worldWidth * (worldHeight - 1) + 128; i < worldWidth * (worldHeight - 1) + 139; i++){
        g_data[i] = 1;
    }
}

static inline void HL_initOnesAtCorners(size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    size_t total_world_size = worldWidth * worldHeight;

    // Initialize the data
    hipMallocManaged(&g_data, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_data, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the resulting data
    hipMallocManaged(&g_resultData, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the above row
    hipMallocManaged(&g_aboveRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_aboveRow, 0, (worldWidth * sizeof(unsigned char)));

    // Initialize the below row
    hipMallocManaged(&g_belowRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_belowRow, 0, (worldWidth * sizeof(unsigned char)));

    if(myrank == 0){
        g_data[0] = 1; // upper left
        g_data[worldWidth-1]=1; // upper right
        g_aboveRow[0] = 1; // upper left
        g_aboveRow[worldWidth-1]=1; // upper right
    }
    if(myrank == cudaDeviceCount - 1){
        g_data[(worldHeight * (worldWidth-1))]=1; // lower left
        g_data[(worldHeight * (worldWidth-1)) + worldWidth-1]=1; // lower right
        g_belowRow[0] = 1;
        g_belowRow[worldWidth - 1] = 1;
    }
}

static inline void HL_initSpinnerAtCorner(size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    size_t total_world_size = worldWidth * worldHeight;

    // Initialize the data
    hipMallocManaged(&g_data, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_data, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the resulting data
    hipMallocManaged(&g_resultData, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the above row
    hipMallocManaged(&g_aboveRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_aboveRow, 0, (worldWidth * sizeof(unsigned char)));

    // Initialize the below row
    hipMallocManaged(&g_belowRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_belowRow, 0, (worldWidth * sizeof(unsigned char)));

    if( myrank == 0 ){
        g_data[0] = 1; // upper left
        g_data[1] = 1; // upper left +1
        g_data[worldWidth-1]=1; // upper right
        
        g_aboveRow[0] = 1; // upper left
        g_aboveRow[1] = 1; // upper left +1
        g_aboveRow[worldWidth-1]=1; // upper right
    }
}

static inline void HL_initReplicator(size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    size_t total_world_size = worldWidth * worldHeight;

    // Initialize the data
    hipMallocManaged(&g_data, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_data, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the resulting data
    hipMallocManaged(&g_resultData, (total_world_size * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (total_world_size * sizeof(unsigned char)));

    // Initialize the above row
    hipMallocManaged(&g_aboveRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_aboveRow, 0, (worldWidth * sizeof(unsigned char)));

    // Initialize the below row
    hipMallocManaged(&g_belowRow, (worldWidth * sizeof(unsigned char)));
    hipMemset(g_belowRow, 0, (worldWidth * sizeof(unsigned char)));

    size_t x, y;
    x = worldWidth/2;
    y = worldHeight/2;
    
    g_data[x + y*worldWidth + 1] = 1; 
    g_data[x + y*worldWidth + 2] = 1;
    g_data[x + y*worldWidth + 3] = 1;
    g_data[x + (y+1)*worldWidth] = 1;
    g_data[x + (y+2)*worldWidth] = 1;
    g_data[x + (y+3)*worldWidth] = 1; 
}

// ---------- EXPORT TO APPROPRIATE COMPILER ---------- //
extern "C" void HL_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    
    // INITIALIZE THE CUDA WORLD
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
    exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
    printf(" Unable to have myrank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
    exit(-1); 
    }

    // INITIALIZE THE PATTERN
    switch(pattern)
    {
    case 0:
	HL_initAllZeros( worldWidth, worldHeight, myrank, cudaDeviceCount );
	break;
	
    case 1:
	HL_initAllOnes( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;
	
    case 2:
	HL_initOnesInMiddle( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;
	
    case 3:
	HL_initOnesAtCorners( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;

    case 4:
	HL_initSpinnerAtCorner( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;

    case 5:
	HL_initReplicator( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;
	
    default:
	printf("Pattern %u has not been implemented \n", pattern);
	exit(-1);
    }
}

// MAIN KERNEL FUNCTION THAT DOES ALL OF THE WORK
__global__ void HL_kernel( unsigned char* d_data, unsigned char* d_resultData, unsigned char* d_aboveRow, unsigned char* d_belowRow, unsigned int worldWidth, unsigned int worldHeight){
    
    // Store index value
    size_t index;

    // Loop over the threads
    for(index = blockIdx.x * blockDim.x + threadIdx.x; index < worldWidth*worldHeight; index += blockDim.x * gridDim.x){

        // Allocate space
        int y0 = ((index + worldHeight - 1) % worldHeight) * worldWidth;
        int y1 = index * worldWidth;
        int y2 = ((index + 1) % worldHeight) * worldWidth;

        // Get the current block and thread
        int x;

        // Loop over corresponding COLUMNS
	        for (x = 0; x < worldWidth; ++x){

            // Set current column, left column, and right column
            int x1 = x;
            int x0 = (x1 + worldWidth - 1) % worldWidth; 
            int x2 = (x1 + 1) % worldWidth;

            // Get the status of the current cell to determine logic of life span
            int is_alive = d_data[x1+y1];

            // Count the number of alive neighbors
            int num_alive = 0;

            // Check above and below row cases
            if (x1+y1 < worldWidth) {
                num_alive = d_aboveRow[x0] + d_aboveRow[x1] + d_aboveRow[x2] + d_data[x0+y1] + d_data[x2+y1] + d_data[x0+y2] + d_data[x1+y2] + d_data[x2+y2];
            }
            else if (x1+y1 > worldWidth*worldHeight - worldWidth - 1) {
                num_alive = d_data[x0+y0] + d_data[x1+y0] + d_data[x2+y0] + d_data[x0+y1] + d_data[x2+y1] + d_belowRow[x0] + d_belowRow[x1] + d_belowRow[x2];
            }
            else {
                num_alive = d_data[x0+y0] + d_data[x1+y0] + d_data[x2+y0] + d_data[x0+y1] + d_data[x2+y1] + d_data[x0+y2] + d_data[x1+y2] + d_data[x2+y2];
            }

            // Logic for updating values
            if (is_alive == 1){
                // Cell is alive!
                if (num_alive < 2){
                    // Underpopulated
                    d_resultData[x1+y1] = 0;
                }
                else if (num_alive == 2 || num_alive == 3){
                    // Just the right amount of neighbors
                    d_resultData[x1+y1] = 1;
                }
                else {
                    // Overpopulated
                    d_resultData[x1+y1] = 0;
                }
            }
            else {
                // Cell is dead :(
                if (num_alive == 3 || num_alive == 6) {
                    // #Resurrected
                    d_resultData[x1+y1] = 1;
                }
                else {
                    // We stay dead
                    d_resultData[x1+y1] = 0;
                }
            }// End logic for staying dead
	    } // End x loop
    } // End loop over each thread

    // ----- SWAP DATA IN ABOVE ROWS AND BELOW ROWS ----- //
    int j;
    for(j = 0; j < worldWidth; j++){
        d_aboveRow[j] = d_resultData[j];
        d_belowRow[j] = d_resultData[j + worldWidth*(worldHeight - 1)];
    }

    // Synchronize the threads?
    __syncthreads();
}

// LAUNCH KERNEL FUNCTION
extern "C" void HL_kernelLaunch( unsigned char** d_data, unsigned char** d_resultData, unsigned char** d_aboveRow, unsigned char** d_belowRow, int block_count, int thread_count, unsigned int worldWidth, unsigned int worldHeight, int myrank){
    
    // Call the kernel
    HL_kernel<<<block_count,thread_count>>>(*d_data, *d_resultData, *d_aboveRow, *d_belowRow, worldWidth, worldHeight);

    // Synchronize the CUDA devices
    hipDeviceSynchronize();
}

// Free memory
extern "C" void freeCudaArrays(int myrank){
    hipFree(g_data);
    hipFree(g_resultData);
    hipFree(g_aboveRow);
    hipFree(g_belowRow);
}
