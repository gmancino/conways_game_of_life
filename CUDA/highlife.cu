// Include packages and also CUDA packages
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>


// Result from last compute of world.
unsigned char *g_resultData=NULL;

// Current state of world. 
unsigned char *g_data=NULL;

// Current width of world.
size_t g_worldWidth=0;

/// Current height of world.
size_t g_worldHeight=0;

/// Current data length (product of width and height)
size_t g_dataLength=0;  // g_worldWidth * g_worldHeight

static inline void HL_initAllZeros( size_t worldWidth, size_t worldHeight )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // g_data = calloc( g_dataLength, sizeof(unsigned char));
    // g_resultData = calloc( g_dataLength, sizeof(unsigned char)); 

    // Set memory to CUDA
    hipMallocManaged(&g_data, (g_dataLength * sizeof(unsigned char)));
    // Zero out the elements
    hipMemset(g_data, 0, (g_dataLength * sizeof(unsigned char)));

    // Same for results
    hipMallocManaged(&g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (g_dataLength * sizeof(unsigned char)));
}

static inline void HL_initAllOnes( size_t worldWidth, size_t worldHeight )
{
    int i;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    // Set memory to CUDA
    hipMallocManaged(&g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, (g_dataLength * sizeof(unsigned char)));

    // set all rows of world to true
    for( i = 0; i < g_dataLength; i++)
    {
	g_data[i] = 1;
    }
    
    hipMallocManaged(&g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (g_dataLength * sizeof(unsigned char)));
}

static inline void HL_initOnesInMiddle( size_t worldWidth, size_t worldHeight )
{
    int i;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged(&g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, (g_dataLength * sizeof(unsigned char)));

    // set first 1 rows of world to true
    for( i = 10*g_worldWidth; i < 11*g_worldWidth; i++)
    {
	if( (i >= ( 10*g_worldWidth + 10)) && (i < (10*g_worldWidth + 20)))
	{
	    g_data[i] = 1;
	}
    }
    
    hipMallocManaged(&g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (g_dataLength * sizeof(unsigned char)));
}

static inline void HL_initOnesAtCorners( size_t worldWidth, size_t worldHeight )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged(&g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, (g_dataLength * sizeof(unsigned char)));

    g_data[0] = 1; // upper left
    g_data[worldWidth-1]=1; // upper right
    g_data[(worldHeight * (worldWidth-1))]=1; // lower left
    g_data[(worldHeight * (worldWidth-1)) + worldWidth-1]=1; // lower right
    
    hipMallocManaged(&g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (g_dataLength * sizeof(unsigned char)));
}

static inline void HL_initSpinnerAtCorner( size_t worldWidth, size_t worldHeight )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged(&g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, (g_dataLength * sizeof(unsigned char)));

    g_data[0] = 1; // upper left
    g_data[1] = 1; // upper left +1
    g_data[worldWidth-1]=1; // upper right
    
    hipMallocManaged(&g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (g_dataLength * sizeof(unsigned char)));
}

static inline void HL_initReplicator( size_t worldWidth, size_t worldHeight )
{
    size_t x, y;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged(&g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, (g_dataLength * sizeof(unsigned char)));

    x = worldWidth/2;
    y = worldHeight/2;
    
    g_data[x + y*worldWidth + 1] = 1; 
    g_data[x + y*worldWidth + 2] = 1;
    g_data[x + y*worldWidth + 3] = 1;
    g_data[x + (y+1)*worldWidth] = 1;
    g_data[x + (y+2)*worldWidth] = 1;
    g_data[x + (y+3)*worldWidth] = 1; 
    
    hipMallocManaged(&g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, (g_dataLength * sizeof(unsigned char))); 
}

static inline void HL_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight )
{
    switch(pattern)
    {
    case 0:
	HL_initAllZeros( worldWidth, worldHeight );
	break;
	
    case 1:
	HL_initAllOnes( worldWidth, worldHeight );
	break;
	
    case 2:
	HL_initOnesInMiddle( worldWidth, worldHeight );
	break;
	
    case 3:
	HL_initOnesAtCorners( worldWidth, worldHeight );
	break;

    case 4:
	HL_initSpinnerAtCorner( worldWidth, worldHeight );
	break;

    case 5:
	HL_initReplicator( worldWidth, worldHeight );
	break;
	
    default:
	printf("Pattern %u has not been implemented \n", pattern);
	exit(-1);
    }
}

static inline void HL_swap( unsigned char **pA, unsigned char **pB)
{
    // Create temporay holder to hold A's values
    unsigned char *temporary;
    temporary = *pA;

    // Perform the swap
    *pA = *pB;
    *pB = temporary;
}
 
/*
// Don't Modify this function or your submitty autograding will not work
static inline void HL_printWorld(size_t iteration)
{
    int i, j;

    printf("Print World - Iteration %lu \n", iteration);
    
    for( i = 0; i < g_worldHeight; i++)
    {
	printf("Row %2d: ", i);
	for( j = 0; j < g_worldWidth; j++)
	{
	    printf("%u ", (unsigned int)g_data[(i*g_worldWidth) + j]);
	}
	printf("\n");
    }

    printf("\n\n");
}*/

// MAIN KERNEL FUNCTION THAT DOES ALL OF THE WORK
__global__ void HL_kernel(unsigned char* d_data, unsigned int worldWidth, unsigned int worldHeight, unsigned char* d_resultData){
    // Store index value
    size_t index;

    // Loop over the threads
    for(index = blockIdx.x * blockDim.x + threadIdx.x; index < worldWidth*worldHeight; index += blockDim.x * gridDim.x){

        // Grab the current y
        int y0 = ((index + worldHeight - 1) % worldHeight) * worldWidth;
        int y1 = index * worldWidth;
        int y2 = ((index + 1) % worldHeight) * worldWidth;

        // Get the current block and thread
        int x;

        // Loop over corresponding COLUMNS
	        for (x = 0; x < worldWidth; ++x){

            // Set current column, left column, and right column
            int x1 = x;
            int x0 = (x1 + worldWidth - 1) % worldWidth; 
            int x2 = (x1 + 1) % worldWidth;

            // Get the status of the current cell to determine logic of life span
            int is_alive = d_data[x1+y1];

            // Count the number of alive neighbors
            int num_alive = 0;
            num_alive = d_data[x0+y0] + d_data[x1+y0] + d_data[x2+y0] + d_data[x0+y1] + d_data[x2+y1] + d_data[x0+y2] + d_data[x1+y2] + d_data[x2+y2];

            // Logic for updating values
            if (is_alive == 1){
                // Cell is alive!
                if (num_alive < 2){
                    // Underpopulated
                    d_resultData[x1+y1] = 0;
                }
                else if (num_alive == 2 || num_alive == 3){
                    // Just the right amount of neighbors
                    d_resultData[x1+y1] = 1;
                }
                else {
                    // Overpopulated
                    d_resultData[x1+y1] = 0;
                }
            }
            else {
                // Cell is dead :(
                if (num_alive == 3 || num_alive == 6) {
                    // #Resurrected
                    d_resultData[x1+y1] = 1;
                }
                else {
                    // We stay dead
                    d_resultData[x1+y1] = 0;
                }
            }
	    } // End x loop

    } // End loop over each thread

    // Synchronize the threads?
    __syncthreads();
}

// LAUNCH KERNEL FUNCTION
bool HL_kernelLaunch(unsigned char** d_data, unsigned char** d_resultData, size_t worldWidth, size_t worldHeight, size_t iterationsCount, ushort threadsCount){
    // Delcare iteration variable
    int i;

    // Declare number of blocks
    int block_count = (worldHeight * worldWidth) / threadsCount;

    // Loop over the iterations
    for(i = 0; i < iterationsCount; i++){

        // Perform kernel operations in parallel over the threads
        HL_kernel<<<block_count,threadsCount>>>(*d_data, worldWidth, worldHeight, *d_resultData);

        // Synchronize the CUDA devices
        hipDeviceSynchronize();

        // Swap the pointers
        HL_swap(d_data, d_resultData);

    } // End iterations loop

    // Synchronize the device again?
    hipDeviceSynchronize();

    return true;
}

int main(int argc, char *argv[])
{
    unsigned int pattern = 0;
    unsigned int worldSize = 0;
    unsigned int iterations = 0;
    unsigned int thread_count = 0;

    printf("This is the HighLife running in Parallel on a GPU.\n");

    if( argc != 5 )
    {
	printf("HighLife requires 4 arguments, 1st is pattern number, 2nd the sq size of the world, 3rd is the number of iterations, and 4th is the thread count, e.g. ./highlife 0 64 2 32 \n");
	exit(-1);
    }

    // Read in arguments
    pattern = atoi(argv[1]);
    worldSize = atoi(argv[2]);
    iterations = atoi(argv[3]);
    thread_count = atoi(argv[4]);
    
    // Initialize the world
    HL_initMaster(pattern, worldSize, worldSize);

    // Launch the kernel
    HL_kernelLaunch(&g_data, &g_resultData, worldSize, worldSize, iterations, thread_count);

    // Free memory
    hipFree(g_data);
    hipFree(g_resultData);
    
    return true;
}
